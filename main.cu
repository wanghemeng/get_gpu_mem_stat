
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <nvml.h>

int main() {
    nvmlReturn_t result;
    nvmlDevice_t device;

    // Initialize NVML library
    result = nvmlInit();
    if (result != NVML_SUCCESS) {
        printf("Failed to initialize NVML library: %s\n", nvmlErrorString(result));
        return 1;
    }

    // Get the first available GPU device
    result = nvmlDeviceGetHandleByIndex(0, &device);
    if (result != NVML_SUCCESS) {
        printf("Failed to get GPU device handle: %s\n", nvmlErrorString(result));
        nvmlShutdown();
        return 1;
    }

    nvmlMemory_t memory;

    // Continuously monitor GPU memory usage
    unsigned int maxMemoryUsageMb = 0;
    while (1) {
        // Get the current memory usage in bytes
        result = nvmlDeviceGetMemoryInfo(device, &memory);
        if (result != NVML_SUCCESS) {
            printf("Failed to get memory info: %s\n", nvmlErrorString(result));
            nvmlShutdown();
            return 1;
        }

        // Convert memory usage to megabytes
        unsigned int memoryUsageMb = memory.used / 1024 / 1024;

        // Update max memory usage if necessary
        if (memoryUsageMb > maxMemoryUsageMb) {
            maxMemoryUsageMb = memoryUsageMb;
        }

        // Print the memory usage in megabytes
        printf("Peak GPU Memory Usage: %u MB\n", maxMemoryUsageMb);

    }

    // Shutdown NVML library
    nvmlShutdown();

    return 0;
}